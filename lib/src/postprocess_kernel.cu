#include "hip/hip_runtime.h"
// Copyright 2022 TIER IV, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <circle_nms_kernel.hpp>
#include <postprocess_kernel.hpp>

#include <thrust/count.h>
#include <thrust/sort.h>

namespace
{
const std::size_t THREADS_PER_BLOCK = 32;
}  // namespace

namespace centerpoint
{

struct is_score_greater
{
  is_score_greater(float t) : t_(t) {}

  __device__ bool operator()(const Box3D & b) { return b.score > t_; }

private:
  float t_{0.0};
};

struct is_kept
{
  __device__ bool operator()(const bool keep) { return keep; }
};

struct score_greater
{
  __device__ bool operator()(const Box3D & lb, const Box3D & rb) { return lb.score > rb.score; }
};

__device__ inline float sigmoid(float x) { return 1.0f / expf(-x); }

// __global__ void generateBoxes3D_kernel(
//   const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
//   const float * out_rot, const float * out_vel, const float voxel_size_x, const float voxel_size_y,
//   const float range_min_x, const float range_min_y, const std::size_t down_grid_size_x,
//   const std::size_t down_grid_size_y, const std::size_t downsample_factor, const int num_class,
//   Box3D * det_boxes3d)
// {
//   // generate boxes3d from the outputs of the network.
//   // shape of out_*: (N, DOWN_GRID_SIZE_Y, DOWN_GRID_SIZE_X)
//   // heatmap: N = num_class, offset: N = 2, z: N = 1, dim: N = 3, rot: N = 2, vel: N = 2
//   const auto yi = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
//   const auto xi = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
//   const auto idx = down_grid_size_x * yi + xi;
//   const auto down_grid_size = down_grid_size_y * down_grid_size_x;

//   if (yi >= down_grid_size_y || xi >= down_grid_size_x) {
//     return;
//   }

//   int label = -1;
//   float max_score = -1;
//   for (int ci = 0; ci < num_class; ci++) {
//     float score = sigmoid(out_heatmap[down_grid_size * ci + idx]);
//     if (score > max_score) {
//       label = ci;
//       max_score = score;
//     }
//   }

//   const float offset_x = out_offset[down_grid_size * 0 + idx];
//   const float offset_y = out_offset[down_grid_size * 1 + idx];
//   const float x = voxel_size_x * downsample_factor * (xi + offset_x) + range_min_x;
//   const float y = voxel_size_y * downsample_factor * (yi + offset_y) + range_min_y;
//   const float z = out_z[idx];
//   const float w = out_dim[down_grid_size * 0 + idx];
//   const float l = out_dim[down_grid_size * 1 + idx];
//   const float h = out_dim[down_grid_size * 2 + idx];
//   const float yaw_sin = out_rot[down_grid_size * 0 + idx];
//   const float yaw_cos = out_rot[down_grid_size * 1 + idx];
//   const float vel_x = out_vel[down_grid_size * 0 + idx];
//   const float vel_y = out_vel[down_grid_size * 1 + idx];

//   det_boxes3d[idx].label = label;
//   det_boxes3d[idx].score = max_score;
//   det_boxes3d[idx].x = x;
//   det_boxes3d[idx].y = y;
//   det_boxes3d[idx].z = z;
//   det_boxes3d[idx].length = expf(l);
//   det_boxes3d[idx].width = expf(w);
//   det_boxes3d[idx].height = expf(h);
//   det_boxes3d[idx].yaw = atan2f(yaw_sin, yaw_cos);
//   det_boxes3d[idx].vel_x = vel_x;
//   det_boxes3d[idx].vel_y = vel_y;
// }
__global__ void generateBoxes3D_kernel(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, const float voxel_size_x, const float voxel_size_y,
  const float range_min_x, const float range_min_y, const std::size_t down_grid_size_x,
  const std::size_t down_grid_size_y, const std::size_t downsample_factor, const int num_class,
  Box3D * det_boxes3d)
{
  // generate boxes3d from the outputs of the network.
  // shape of out_*: (N, DOWN_GRID_SIZE_Y, DOWN_GRID_SIZE_X)
  // heatmap: N = num_class, offset: N = 2, z: N = 1, dim: N = 3, rot: N = 2, vel: N = 2
  const auto yi = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
  const auto xi = blockIdx.y * THREADS_PER_BLOCK + threadIdx.y;
  const auto idx = down_grid_size_x * yi + xi;
  const auto down_grid_size = down_grid_size_y * down_grid_size_x;

  if (yi >= down_grid_size_y || xi >= down_grid_size_x) {
    return;
  }

  int label = -1;
  float max_score = -1;
  for (int ci = 0; ci < num_class; ci++) {
    float score = sigmoid(out_heatmap[down_grid_size * ci + idx]);
    if (score > max_score) {
      label = ci;
      max_score = score;
    }
  }

  const float offset_x = out_offset[down_grid_size * 0 + idx];
  const float offset_y = out_offset[down_grid_size * 1 + idx];
  const float x = voxel_size_x * downsample_factor * (xi + offset_x) + range_min_x;
  const float y = voxel_size_y * downsample_factor * (yi + offset_y) + range_min_y;
  const float z = out_z[idx];
  // const float w = out_dim[down_grid_size * 0 + idx];
  // const float l = out_dim[down_grid_size * 1 + idx];
  // const float h = out_dim[down_grid_size * 2 + idx]; //mmdet3d dim[w,l,h]
  const float l = out_dim[down_grid_size * 0 + idx];
  const float w = out_dim[down_grid_size * 1 + idx];
  const float h = out_dim[down_grid_size * 2 + idx]; //openPCDet dim[l,w,h]
  // const float yaw_sin = out_rot[down_grid_size * 0 + idx];
  // const float yaw_cos = out_rot[down_grid_size * 1 + idx];//mmdet3d[sin,cos]
  const float yaw_cos = out_rot[down_grid_size * 0 + idx];
  const float yaw_sin = out_rot[down_grid_size * 1 + idx];//openPCDet [cos,sin]
  // const float vel_x = out_vel[down_grid_size * 0 + idx];
  // const float vel_y = out_vel[down_grid_size * 1 + idx];

  det_boxes3d[idx].label = label;
  det_boxes3d[idx].score = max_score;
  det_boxes3d[idx].x = x;
  det_boxes3d[idx].y = y;
  det_boxes3d[idx].z = z;
  det_boxes3d[idx].length = expf(l);
  det_boxes3d[idx].width = expf(w);
  det_boxes3d[idx].height = expf(h);
  det_boxes3d[idx].yaw = atan2f(yaw_sin, yaw_cos);
  // det_boxes3d[idx].vel_x = vel_x;
  // det_boxes3d[idx].vel_y = vel_y;
}

PostProcessCUDA::PostProcessCUDA(const std::size_t num_class, const float score_threshold)
: num_class_(num_class), score_threshold_(score_threshold)
{
  const auto num_raw_boxes3d = Config::down_grid_size_y * Config::down_grid_size_x;
  boxes3d_d_ = thrust::device_vector<Box3D>(num_raw_boxes3d);
}

// hipError_t PostProcessCUDA::generateDetectedBoxes3D_launch(
//   const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
//   const float * out_rot, const float * out_vel, std::vector<Box3D> & det_boxes3d,
//   hipStream_t stream)
// {
//   dim3 blocks(
//     divup(Config::down_grid_size_y, THREADS_PER_BLOCK),
//     divup(Config::down_grid_size_x, THREADS_PER_BLOCK));
//   dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
//   generateBoxes3D_kernel<<<blocks, threads, 0, stream>>>(
//     out_heatmap, out_offset, out_z, out_dim, out_rot, out_vel, Config::voxel_size_x,
//     Config::voxel_size_y, Config::range_min_x, Config::range_min_y, Config::down_grid_size_x,
//     Config::down_grid_size_y, Config::downsample_factor, num_class_,
//     thrust::raw_pointer_cast(boxes3d_d_.data()));

//   // suppress by socre
//   const auto num_det_boxes3d = thrust::count_if(
//     thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(), is_score_greater(score_threshold_));
//   if (num_det_boxes3d == 0) {
//     return hipGetLastError();
//   }
//   thrust::device_vector<Box3D> det_boxes3d_d(num_det_boxes3d);
//   thrust::copy_if(
//     thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(), det_boxes3d_d.begin(),
//     is_score_greater(score_threshold_));

//   // sort by score
//   thrust::sort(det_boxes3d_d.begin(), det_boxes3d_d.end(), score_greater());

//   // supress by NMS
//   thrust::device_vector<bool> final_keep_mask_d(num_det_boxes3d);
//   const auto num_final_det_boxes3d =
//     circleNMS(det_boxes3d_d, dist_threshold_, final_keep_mask_d, stream);

//   thrust::device_vector<Box3D> final_det_boxes3d_d(num_final_det_boxes3d);
//   thrust::copy_if(
//     thrust::device, det_boxes3d_d.begin(), det_boxes3d_d.end(), final_keep_mask_d.begin(),
//     final_det_boxes3d_d.begin(), is_kept());

//   // memcpy device to host
//   det_boxes3d.resize(num_final_det_boxes3d);
//   thrust::copy(final_det_boxes3d_d.begin(), final_det_boxes3d_d.end(), det_boxes3d.begin());

//   return hipGetLastError();
// }
hipError_t PostProcessCUDA::generateDetectedBoxes3D_launch(
  const float * out_heatmap, const float * out_offset, const float * out_z, const float * out_dim,
  const float * out_rot, std::vector<Box3D> & det_boxes3d,
  hipStream_t stream)
{
  dim3 blocks(
    divup(Config::down_grid_size_y, THREADS_PER_BLOCK),
    divup(Config::down_grid_size_x, THREADS_PER_BLOCK));
  dim3 threads(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
  generateBoxes3D_kernel<<<blocks, threads, 0, stream>>>(
    out_heatmap, out_offset, out_z, out_dim, out_rot, Config::voxel_size_x,
    Config::voxel_size_y, Config::range_min_x, Config::range_min_y, Config::down_grid_size_x,
    Config::down_grid_size_y, Config::downsample_factor, num_class_,
    thrust::raw_pointer_cast(boxes3d_d_.data()));

  // suppress by socre
  const auto num_det_boxes3d = thrust::count_if(
    thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(), is_score_greater(score_threshold_));
  if (num_det_boxes3d == 0) {
    return hipGetLastError();
  }
  thrust::device_vector<Box3D> det_boxes3d_d(num_det_boxes3d);
  thrust::copy_if(
    thrust::device, boxes3d_d_.begin(), boxes3d_d_.end(), det_boxes3d_d.begin(),
    is_score_greater(score_threshold_));

  // sort by score
  thrust::sort(det_boxes3d_d.begin(), det_boxes3d_d.end(), score_greater());

  // supress by NMS
  thrust::device_vector<bool> final_keep_mask_d(num_det_boxes3d);
  const auto num_final_det_boxes3d =
    circleNMS(det_boxes3d_d, dist_threshold_, final_keep_mask_d, stream);

  thrust::device_vector<Box3D> final_det_boxes3d_d(num_final_det_boxes3d);
  thrust::copy_if(
    thrust::device, det_boxes3d_d.begin(), det_boxes3d_d.end(), final_keep_mask_d.begin(),
    final_det_boxes3d_d.begin(), is_kept());

  // memcpy device to host
  det_boxes3d.resize(num_final_det_boxes3d);
  thrust::copy(final_det_boxes3d_d.begin(), final_det_boxes3d_d.end(), det_boxes3d.begin());

  return hipGetLastError();
}

}  // namespace centerpoint
